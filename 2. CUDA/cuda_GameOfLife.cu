#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include "hip/hip_runtime.h"

#include ""
#include "header.cuh"
#include "cuda_GameOfLife.cuh"


int main (int argc, char **argv) {

	/* for Broadcast */

	int width = XDIMENSION;
	int height = XDIMENSION;


	/* Initialize arrays */
	
	char** t;
	char** t1;
	char* gpu_t;
	char* gpu_t1;
	char* gpu_temp;
	

	t = malloc(sizeof(char*)*width);
	t1 = malloc(sizeof(char*)*width);

	for(i = 0; i < width; i++) {
		t[i] = malloc(sizeof(char)*height);
		t1[i] = malloc(sizeof(char)*height);
	}

	hipMalloc(&gpu_t, width*height*sizeof(char));
	hipMalloc(&gpu_t1, width*height*sizeof(char));


	int blocks = BLOCKS; 

	
	/* Initialization of t array */
   
	for(i=0; i<x*x; i++)
		t[i] = (rand()%2 == 0)?DEAD:ALIVE;

	hipMemcpy(gpu_t, t, width*height, hipMemcpyHostToDevice);

	hipMemcpy(gpu_t1, t1, width*height, hipMemcpyHostToDevice);


	/* Update the grid STEP times */

	for (timer = 0 ; timer<GENERATIONS; timer++) {
		dim3 blockSize(blocks, blocks);
		dim3 dimGrid(width, heigt);

		nextGenerationCells<<<blockSize, dimGrid>>>(gpu_t, gpu_t1, width, height);

		/* Swap arrays */
		gpu_temp = gpu_t;
		gpu_t = gpu_t1;
		gpu_t1 = gpu_temp;

		if(!isDifferent(gpu_t, gpu_t1, height, width))
			break;
	}

	hipMemcpy(t, gpu_t, width*height, hipMemcpyDeviceToHost);
	hipMemcpy(t1, gpu_t1, width*height, hipMemcpyDeviceToHost);



	/* Print final grid at file */
	
	printData(height, width, t1, "final.txt");
	
	
	/* Free allocated space */
	for(i = 0; i < width; i++){
		free(t[i]);
		free(t1[i]);
	}
	free(t);
	free(t1);
	hipFree(gpu_t);
	hipFree(gpu_t1);
	
	return 0;

}


