#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

#include "cuda_GameOfLife.cuh"

/**************************************************************************
	.validateInput
****************************************************************************/


void validateInput(int numOfWorkers) {		// not touched yet

	//Simplifying the problem
	//Demanding that number of workers is a square number whose square root is perfectly divided by Grid's X & Y dimensions
	//Plus 1 task -> MASTER

	int sqrtWorkers = sqrt(numOfWorkers);
  	if ((numOfWorkers > MAXWORKER) || (numOfWorkers < MINWORKER) || sqrtWorkers*sqrtWorkers != numOfWorkers  || XDIMENSION%sqrtWorkers != 0) {
    	printf("\n\n ERROR: The number of tasks must be between %d and %d.\n 	number of WORKERS must be a square number whose square root is perfectly divided by %d.\n	Total #tasks must be #workers+1.\n", MINWORKER+1,MAXWORKER, XDIMENSION);
    	printf(" Quitting...\n\n");
	    MPI_Abort(MPI_COMM_WORLD, 0);
	    exit(1);
    }
}


/**************************************************************************
	.getRestNeighbors
****************************************************************************/

void getRestNeighbors(MPI_Comm cartesianComm, int *myCoords, int sqrtWorkers, Neighborhood *neighbors){	// not touched yet

	int temp[2], rank, east;
	int x = myCoords[0], y = myCoords[1];
	
	temp[0] = (x!=0) ? x-1 : sqrtWorkers-1;
	
	temp[1] = (y!=sqrtWorkers-1) ? y+1 : 0;						//NorthEast
	east = temp[1];
	MPI_Cart_rank(cartesianComm, temp, &rank);
	neighbors->northeast = rank;
						
	temp[1] = (y!=0) ? y-1 : sqrtWorkers-1;						//NorthWest
	MPI_Cart_rank(cartesianComm, temp, &rank);
	neighbors->northwest = rank;

/*************************************************/

	temp[0] = (x!=sqrtWorkers-1) ? x+1 : 0;

	MPI_Cart_rank(cartesianComm, temp, &rank);					//SouthWest
	neighbors->southwest = rank;

	temp[1] = east;												//SouthEast
	MPI_Cart_rank(cartesianComm, temp, &rank);
	neighbors->southeast = rank;
}


/**************************************************************************
	.nextGenerationInsideCells
****************************************************************************/
__global__ void nextGenerationInsideCells(char *t, char *t1, int width, int height) {

	int i, j, k, aliveCounter=0;
	
	size_t x = blockIdx.x * blockIdDim.x + threadIdx.x; 
	size_t y = blockIdx.y * blockIdDim.y + threadIdx.y;

	if (0 < x && x < height-1 && 0 < y && y < width-1) {		// if inside cells
		for(i=1; i<height-1; i++) {
			for(j=1; j<width-1; j++) {
				if(t[i-1][j]==ALIVE)      aliveCounter++;         // north neighbor
				if(t[i][j+1]==ALIVE)      aliveCounter++;         // east neighbor
				if(t[i+1][j]==ALIVE)      aliveCounter++;         // south neighbor
				if(t[i][j-1]==ALIVE)      aliveCounter++;         // west neighbor
				if(t[i-1][j+1]==ALIVE)    aliveCounter++;         // northeast neighbor
				if(t[i+1][j+1]==ALIVE)    aliveCounter++;         // southeast neighbor
				if(t[i+1][j-1]==ALIVE)    aliveCounter++;         // southwest neighbor
				if(t[i-1][j-1]==ALIVE)    aliveCounter++;         // northwest neighbor
				if(t[i][j]==ALIVE) {
					if(aliveCounter==2 || aliveCounter==3)          t1[i][j]=ALIVE;
					else            t1[i][j]=DEAD;
				}
				else {
					if(aliveCounter==3)             t1[i][j]=ALIVE;
					else            t1[i][j]=DEAD;
				}
				aliveCounter=0;
			}
		}
	}
}


/**************************************************************************
	.nextGenerationOutsideCells
****************************************************************************/
__global__ void nextGenerationOutsideCells(char *t, char *t1, int width, int height,  ReceiveBuffer neighbors){
	int i, j, k, aliveCounter=0;
	
	size_t x = blockIdx.x * blockIdDim.x + threadIdx.x; 
	size_t y = blockIdx.y * blockIdDim.y + threadIdx.y;

	if ((0 == x || x == height-1) && (0 == y || y || width-1)) {		// if outside cells
	for(i=0; i<height; i++) {
		for(j=0; j<width; j++) {
			if(i==0 || i==x-1 || j==0 || j==x-1) {

				// north neighbor
				if(i-1<0) {
					if(neighbors.north[j]==ALIVE)			aliveCounter++;
				}
				else {
					if(fromGrid[i-1][j]==ALIVE)			aliveCounter++;
				}

				// east neighbor
				if(j+1>=x) {
					if(neighbors.east[i]==ALIVE)			aliveCounter++;
				}
				else {
					if(fromGrid[i][j+1]==ALIVE)			aliveCounter++;
				}

				// south neighbor
				if(i+1>=x) {
					if(neighbors.south[j]==ALIVE)			aliveCounter++;
				}
				else {
					if(fromGrid[i+1][j]==ALIVE)			aliveCounter++;
				}

				// west neighbor
				if(j-1<0) {
					if(neighbors.west[i]==ALIVE)			aliveCounter++;
				}
				else {
					if(fromGrid[i][j-1]==ALIVE)			aliveCounter++;
				}
			
				// northeast neighbor
				if(i==0 && j==x-1) {
					if(neighbors.northeast==ALIVE)		aliveCounter++;
				}
				else if(i==0 && j!=x-1) {
					if(neighbors.north[j+1]==ALIVE)		aliveCounter++;
				}
				else if(i!=0 && j==x-1) {
					if(neighbors.east[i-1]==ALIVE)		aliveCounter++;
				}
				else {
					if(fromGrid[i-1][j+1]==ALIVE)			aliveCounter++;
				}

				// southeast neighbor
				if(i!=x-1 && j==x-1) {
					if(neighbors.east[i+1]==ALIVE)		aliveCounter++;
				}
				else if(i==x-1 && j==x-1) {
					if(neighbors.southeast==ALIVE)		aliveCounter++;
				}
				else if(i==x-1 && j!=x-1) {
					if(neighbors.south[j+1]==ALIVE)		aliveCounter++;
				}
				else {
					if(fromGrid[i+1][j+1]==ALIVE)	aliveCounter++;
				}

				// southwest neighbor
				if(i!=x-1 && j==0) {
					if(neighbors.west[i+1]==ALIVE)		aliveCounter++;
				}
				else if(i==x-1 && j==0) {
					if(neighbors.southwest==ALIVE)		aliveCounter++;
				}
				else if(i==x-1 && j!=0) {
					if(neighbors.south[j-1]==ALIVE)		aliveCounter++;
				}
				else {
					if(fromGrid[i+1][j-1]==ALIVE)			aliveCounter++;
				}

				// northwest neighbor
				if(i==0 && j==0) {
					if(neighbors.northwest==ALIVE)		aliveCounter++;
				}
				else if(i==0 && j!=0) {
					if(neighbors.north[j-1]==ALIVE)		aliveCounter++;
				}
				else if(i!=0 && j==0) {
					if(neighbors.west[i-1]==ALIVE)		aliveCounter++;
				}
				else {
					if(fromGrid[i-1][j-1]==ALIVE)			aliveCounter++;
				}


				// Check if it will be alive or dead at next generation
				if(fromGrid[i][j]==ALIVE) {
					if(aliveCounter==2 || aliveCounter==3)		toGrid[i][j]=ALIVE;
					else		toGrid[i][j]=DEAD;
				}
				else {
					if(aliveCounter==3)		toGrid[i][j]=ALIVE;
					else		toGrid[i][j]=DEAD;
				}
				aliveCounter=0;
			}
		}
	}
}


/**************************************************************************
	.isDifferent
**************************************************************************/
int isDifferent(char *t, char *t1, int x) {													// needs changes

	int i;
	for(i=0; i<x*x; i++){
		if(t[i] != t1[i])
			return 1;
	}

	return 0;
}


/**************************************************************************
	.printData
**************************************************************************/
void printData(int x, int sqrtWorkers, char ***subarraysptr, char* fnam) {				// needs changes

	FILE *fp;
	int i, j, z, subline;
	char *subarray;

	fp = fopen(fnam, "w");

	for(i=0; i<sqrtWorkers; i++){					//subarray of task (i,j)
		for(subline=0; subline<x; subline++){		//subline line of subarray
			for(j=0; j<sqrtWorkers; j++){
				subarray = subarraysptr[i][j];
				for(z=0; z<x; z++){
					fprintf(fp, "%c ", subarray[subline*x+z]);	
				}
			}		
			fprintf(fp, "\n");
		}
	}
	fclose(fp);
}
